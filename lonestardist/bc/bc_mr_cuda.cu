#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"
#include "hipcub/hipcub.hpp"
#include "cub/util_allocator.cuh"
#include "thread_work.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);

// thread block size
#define TB_SIZE 256

//#include "kernels/reduce.cuh"
#include "bc_mr_cuda.cuh"

#include "mrbc_tree_cuda.cuh"

// TODO: WESTON: remove bcData variables

//   galois::do_all(
//       galois::iterate(allNodes.begin(), allNodes.end()),
//       [&](GNode curNode) {
//         NodeData& cur_data = graph.getData(curNode);
//         cur_data.sourceData.resize(vectorSize);
//         cur_data.bc = 0.0;
//       },  

__global__ void InitializeGraph_kernel(
        CSRGraph graph, 
        unsigned int __begin, 
        unsigned int __end, 
        float * p_bc)
 {
   unsigned tid = TID_1D;
   unsigned nthreads = TOTAL_THREADS_1D;

   //const unsigned __kernel_tb_size = TB_SIZE;
   index_type src_end =  __end;
   for (index_type src = __begin + tid; src < src_end; src += nthreads)
   {
       p_bc[src] = 0;
   }
 }

void InitializeGraph_allNodes_cuda(struct CUDA_Context* ctx) {
  dim3 blocks;
  dim3 threads;

  kernel_sizing(blocks, threads);
  
  InitializeGraph_kernel <<<blocks, threads>>>(
          ctx->gg, 
          0, 
          ctx->gg.nnodes, 
          ctx->bc.data.gpu_wr_ptr());

  hipDeviceSynchronize();
  check_cuda_kernel;
}



//   galois::do_all(
//       galois::iterate(allNodes.begin(), allNodes.end()),
//       [&](GNode curNode) {
//         NodeData& cur_data = graph.getData(curNode);
//         cur_data.roundIndexToSend = infinity;
//         cur_data.dTree.initialize();
//         for (unsigned i = 0; i < numSourcesPerRound; i++) {
//           // min distance and short path count setup
//           if (nodesToConsider[i] == graph.getGID(curNode)) { // source node
//             cur_data.sourceData[i].minDistance = 0;
//             cur_data.sourceData[i].shortPathCount = 1;
//             cur_data.sourceData[i].dependencyValue = 0.0;
//             cur_data.dTree.setDistance(i, 0);
//           } else { // non-source node
//             cur_data.sourceData[i].minDistance = infinity;
//             cur_data.sourceData[i].shortPathCount = 0;
//             cur_data.sourceData[i].dependencyValue = 0.0;
//           }
//         }
//       },
//       galois::loopname(syncSubstrate->get_run_identifier("InitializeIteration").c_str()),
//       galois::no_stats());

__global__ void InitializeIteration_kernel(
        CSRGraph graph, 
        unsigned int __begin, 
        unsigned int __end, 
        uint64_t *  cuda_nodes_to_consider,
        uint32_t local_infinity,
        unsigned int numSourcesPerRound,
        uint32_t * p_minDistance,
        double   * p_shortPathCount,
        float    * p_dependencyValue,
        uint32_t * p_roundIndexToSend,
        MRBCTree_cuda * p_mrbc_tree)
 {
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  //const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads) {
    p_roundIndexToSend[src] = local_infinity;
    p_mrbc_tree[src].initialize();
    for (index_type i = 0; i < numSourcesPerRound; i++) {
      unsigned int index = src + (i * graph.nnodes);
      if (graph.node_data[src] == cuda_nodes_to_consider[i]) {
        p_minDistance[index] = 0;
        p_shortPathCount[index] = 1;
        p_dependencyValue[index] = 0.0;
        p_mrbc_tree[src].setDistance(0, 0);
       } else {
         p_minDistance[index] = local_infinity;
         p_shortPathCount[index] = 0;
         p_dependencyValue[index] = 0.0;
       }
     }
  }
}

void InitializeIteration_allNodes_cuda(
    const uint32_t & local_infinity, 
    const uint64_t* local_nodes_to_consider, 
    struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;

  kernel_sizing(blocks, threads);

  // Make device vector for local_nodes_to_consider
  uint64_t* cuda_nodes_to_consider;
  hipMalloc((void**) &cuda_nodes_to_consider, ctx->vectorSize*sizeof(uint64_t));
  hipMemcpy(cuda_nodes_to_consider, local_nodes_to_consider, ctx->vectorSize*sizeof(uint64_t), hipMemcpyHostToDevice);
 
  InitializeIteration_kernel <<<blocks, threads>>>(
          ctx->gg, 
          0, 
          ctx->gg.nnodes, 
          cuda_nodes_to_consider,
          local_infinity,
          ctx->vectorSize,
          ctx->minDistance.data.gpu_wr_ptr(),
          ctx->shortPathCount.data.gpu_wr_ptr(),
          ctx->dependencyValue.data.gpu_wr_ptr(),
          ctx->roundIndexToSend.data.gpu_wr_ptr(),
          ctx->mrbc_tree.data.gpu_wr_ptr());
  hipDeviceSynchronize();
  check_cuda_kernel;
}




//   galois::do_all(
//       galois::iterate(allNodes.begin(), allNodes.end()),
//       [&](GNode curNode) {
//         NodeData& cur_data = graph.getData(curNode);
//         cur_data.roundIndexToSend = cur_data.dTree.getIndexToSend(roundNumber);
// 
//         if (cur_data.roundIndexToSend != infinity) {
//           if (cur_data.sourceData[cur_data.roundIndexToSend].minDistance != 0) {
//             bitset_minDistances.set(curNode);
//           }
//           dga += 1;
//         } else if (cur_data.dTree.moreWork()) {
//           dga += 1;
//         }
//       },

__global__ void FindMessageToSync_kernel(
        CSRGraph graph, 
        unsigned int __begin, 
        unsigned int __end, 
        uint32_t roundNumber,
        uint32_t local_infinity,
        uint32_t * p_minDistance,
        uint32_t * p_roundIndexToSend,
        MRBCTree_cuda * p_mrbc_tree,
        DynamicBitset& bitset_minDistance,
        HGAccumulator<uint32_t> dga)
{
    unsigned tid = TID_1D;
    unsigned nthreads = TOTAL_THREADS_1D;

    //const unsigned __kernel_tb_size = TB_SIZE;
    __shared__ hipcub::BlockReduce<uint32_t, TB_SIZE>::TempStorage dga_ts;
    index_type src_end;

    dga.thread_entry();

    src_end = __end;
    for (index_type src = __begin + tid; src < src_end; src += nthreads)
    {
        p_roundIndexToSend[src] = p_mrbc_tree[src].getIndexToSend(roundNumber);

        if (p_roundIndexToSend[src] != local_infinity) {
            if (p_minDistance[p_roundIndexToSend[src] * __end + src] != 0) {
              bitset_minDistance.set(p_roundIndexToSend[src] * graph.nnodes + src);
            }
            dga.reduce(1);
        } else if ( p_mrbc_tree[src].moreWork() ) {
            dga.reduce(1);
        }

    }

    dga.thread_exit<hipcub::BlockReduce<uint32_t, TB_SIZE> >(dga_ts);
}

void FindMessageToSync_cuda(
    uint32_t roundNumber,
    const uint32_t & local_infinity, 
    uint32_t &dga,
    struct CUDA_Context*  ctx)
{

  dim3 blocks;
  dim3 threads;
  kernel_sizing(blocks, threads);

  HGAccumulator<uint32_t> _dga;
  Shared<uint32_t> dgaval  = Shared<uint32_t>(1);
  *(dgaval.cpu_wr_ptr()) = 0;
  _dga.rv = dgaval.gpu_wr_ptr();

  FindMessageToSync_kernel <<<blocks, threads>>>(
          ctx->gg, 
          0, 
          ctx->gg.nnodes, 
          roundNumber,
          local_infinity,
          ctx->minDistance.data.gpu_rd_ptr(),
          ctx->roundIndexToSend.data.gpu_wr_ptr(),
          ctx->mrbc_tree.data.gpu_wr_ptr(),
          *(ctx->minDistance.is_updated.gpu_wr_ptr()), 
          _dga);

  hipDeviceSynchronize();
  check_cuda_kernel;
  
  dga = *(dgaval.cpu_rd_ptr());
}

//   galois::do_all(
//       galois::iterate(allNodes.begin(), allNodes.end()),
//       [&](GNode curNode) {
//         NodeData& cur_data = graph.getData(curNode);
//         if (cur_data.roundIndexToSend != infinity) {
//           cur_data.dTree.markSent(roundNumber);
//         }
//       },
//       galois::loopname(
//           syncSubstrate->get_run_identifier("ConfirmMessageToSend").c_str()),
//       galois::no_stats());

__global__ void ConfirmMessageToSend_kernel(
        CSRGraph graph, 
        unsigned int __begin, 
        unsigned int __end, 
        uint32_t roundNumber,
        uint32_t local_infinity,
        uint32_t * p_bcData_minDistance,
        double   * p_bcData_shortPathCount,
        float    * p_bcData_dependencyValue,
        uint32_t * p_roundIndexToSend,
        MRBCTree_cuda * p_mrbc_tree)
 {
   unsigned tid = TID_1D;
   unsigned nthreads = TOTAL_THREADS_1D;

   //const unsigned __kernel_tb_size = TB_SIZE;
   index_type src_end;
   src_end = __end;
   for (index_type src = __begin + tid; src < src_end; src += nthreads)
   {
     if (p_roundIndexToSend[src] != local_infinity) {
       p_mrbc_tree[src].markSent(roundNumber);
     }

   }
 }

void ConfirmMessageToSend_cuda(
    uint32_t roundNumber,
    const uint32_t & local_infinity, 
    struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;

  kernel_sizing(blocks, threads);
  
  ConfirmMessageToSend_kernel <<<blocks, threads>>>(
          ctx->gg, 
          0, 
          ctx->gg.nnodes, 
          roundNumber,
          local_infinity,
          ctx->bcData_minDistance.data.gpu_wr_ptr(),
          ctx->bcData_shortPathCount.data.gpu_wr_ptr(),
          ctx->bcData_dependencyValue.data.gpu_wr_ptr(),
          ctx->roundIndexToSend.data.gpu_wr_ptr(),
          ctx->mrbc_tree.data.gpu_wr_ptr());

  hipDeviceSynchronize();
  check_cuda_kernel;
}



__global__ void SendAPSPMessages_kernel(
       CSRGraph graph, 
       unsigned int __begin, 
       unsigned int __end, 
       uint32_t local_infinity,
       uint32_t * p_bcData_minDistance,
       double   * p_bcData_shortPathCount,
       float    * p_bcData_dependencyValue,
       uint32_t * p_roundIndexToSend,
       MRBCTree_cuda * p_mrbc_tree,
       HGAccumulator<uint32_t> dga)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  //const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ hipcub::BlockReduce<uint32_t, TB_SIZE>::TempStorage dga_ts;
  index_type dst_end;

  dga.thread_entry();

  dst_end = __end;
  for (index_type dst = __begin + tid; dst < dst_end; dst += nthreads)
  {

    index_type current_edge_end = graph.getFirstEdge((dst) + 1);
    for (index_type current_edge = graph.getFirstEdge(dst); 
           current_edge < current_edge_end;   
           current_edge += 1)
    {   
        index_type src = graph.getAbsDestination(current_edge);
        uint32_t indexToSend = p_roundIndexToSend[src];
       
        if (indexToSend != local_infinity) {
            uint32_t distValue = p_bcData_minDistance[src];
            uint32_t newValue  = distValue + 1;
            // Update minDistance vector
            uint32_t oldValue = p_bcData_minDistance[dst];

            if (oldValue > newValue) {
                p_bcData_minDistance[dst] = newValue;
                p_mrbc_tree[dst].setDistance(indexToSend, oldValue, newValue);
                p_bcData_shortPathCount[dst] = p_bcData_shortPathCount[src];
            } else if (oldValue == newValue) {
                // assert (p_bcData_shortPathCount[dst]
                // add to short path
                p_bcData_shortPathCount[dst] += p_bcData_shortPathCount[src];
            }

            // dga += 1
            dga.reduce(1);
        }
    }
  }
  dga.thread_exit<hipcub::BlockReduce<uint32_t, TB_SIZE> >(dga_ts);

}





void SendAPAPMessages_cuda(
    const uint32_t & local_infinity, 
    uint32_t &dga,
    struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;

  kernel_sizing(blocks, threads);
  
  HGAccumulator<uint32_t> _dga;
  Shared<uint32_t> dgaval  = Shared<uint32_t>(1);
  *(dgaval.cpu_wr_ptr()) = 0;
  _dga.rv = dgaval.gpu_wr_ptr();

  SendAPSPMessages_kernel <<<blocks, threads>>>(
          ctx->gg, 
          0, 
          ctx->gg.nnodes, 
          local_infinity,
          ctx->bcData_minDistance.data.gpu_wr_ptr(),
          ctx->bcData_shortPathCount.data.gpu_wr_ptr(),
          ctx->bcData_dependencyValue.data.gpu_wr_ptr(),
          // TODO: WESTON: hash map info?
          ctx->roundIndexToSend.data.gpu_wr_ptr(),
          ctx->mrbc_tree.data.gpu_wr_ptr(),
          _dga);

  hipDeviceSynchronize();
  check_cuda_kernel;

  dga = *(dgaval.cpu_rd_ptr());
}


//   galois::do_all(
//       galois::iterate(allNodes.begin(), allNodes.end()),
//       [&](GNode node) {
//         NodeData& cur_data = graph.getData(node);
//         cur_data.dTree.prepForBackPhase();
//       },  
//       galois::loopname(
//           syncSubstrate->get_run_identifier("RoundUpdate").c_str()),
//       galois::no_stats());


__global__ void RoundUpdate_kernel(
       CSRGraph graph, 
       unsigned int __begin, 
       unsigned int __end, 
       uint32_t local_infinity,
       uint32_t * p_bcData_minDistance,
       double   * p_bcData_shortPathCount,
       float    * p_bcData_dependencyValue,
       uint32_t * p_roundIndexToSend,
       MRBCTree_cuda * p_mrbc_tree)
{
    unsigned tid = TID_1D;
    unsigned nthreads = TOTAL_THREADS_1D;

    //const unsigned __kernel_tb_size = TB_SIZE;
    index_type src_end;

    src_end = __end;
    for (index_type src = __begin + tid; src < src_end; src += nthreads)
    {
      p_mrbc_tree[src].prepForBackPhase();
    }

}


void RoundUpdate_cuda(
    const uint32_t & local_infinity, 
    struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;

  kernel_sizing(blocks, threads);
  
  RoundUpdate_kernel <<<blocks, threads>>>(
          ctx->gg, 
          0, 
          ctx->gg.nnodes, 
          local_infinity,
          ctx->bcData_minDistance.data.gpu_wr_ptr(),
          ctx->bcData_shortPathCount.data.gpu_wr_ptr(),
          ctx->bcData_dependencyValue.data.gpu_wr_ptr(),
          // TODO: WESTON: hash map info?
          ctx->roundIndexToSend.data.gpu_wr_ptr(),
          ctx->mrbc_tree.data.gpu_wr_ptr());

  hipDeviceSynchronize();
  check_cuda_kernel;
}



//   galois::do_all(
//       galois::iterate(allNodes.begin(), allNodes.end()),
//       [&](GNode dst) {
//         NodeData& dst_data        = graph.getData(dst);
//
//         // if zero distances already reached, there is no point sending things
//         // out since we don't care about dependecy for sources (i.e. distance
//         // 0)
//         if (!dst_data.dTree.isZeroReached()) {
//           dst_data.roundIndexToSend =
//             dst_data.dTree.backGetIndexToSend(roundNumber, lastRoundNumber);
//
//           if (dst_data.roundIndexToSend != infinity) {
//             // only comm if not redundant 0
//             if (dst_data.sourceData[dst_data.roundIndexToSend].dependencyValue != 0) {
//               bitset_dependency.set(dst);
//             }
//           }
//         }
//       },
//       galois::loopname(
//         syncSubstrate->get_run_identifier("BackFindMessageToSend").c_str()
//       ),
//       galois::no_stats());

__global__ void BackFindMessageToSend_kernel(
       CSRGraph graph, 
       unsigned int __begin, 
       unsigned int __end, 
       uint32_t local_infinity,
       uint32_t roundNumber,
       uint32_t lastRoundNumber,
       uint32_t * p_bcData_minDistance,
       double   * p_bcData_shortPathCount,
       float    * p_bcData_dependencyValue,
       uint32_t * p_roundIndexToSend,
       MRBCTree_cuda * p_mrbc_tree)
{
    unsigned tid = TID_1D;
    unsigned nthreads = TOTAL_THREADS_1D;

    //const unsigned __kernel_tb_size = TB_SIZE;
    index_type src_end;

    src_end = __end;
    for (index_type src = __begin + tid; src < src_end; src += nthreads)
    {
      if (p_mrbc_tree[src].isZeroReached()) {
        p_roundIndexToSend[src] = 
          p_mrbc_tree[src].backGetIndexToSend(roundNumber, lastRoundNumber);

        if (p_roundIndexToSend[src] != local_infinity) {
             // TODO: WESTON: update this with bitset stuff ELENA?
             //if (dst_data.sourceData[dst_data.roundIndexToSend].dependencyValue != 0) {
             //  bitset_dependency.set(dst);
             //}
        }
      }
    }

}


void BackFindMessageToSend_cuda(
    const uint32_t & local_infinity, 
    uint32_t roundNumber,
    uint32_t lastRoundNumber,
    struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;

  kernel_sizing(blocks, threads);
  
  BackFindMessageToSend_kernel <<<blocks, threads>>>(
          ctx->gg, 
          0, 
          ctx->gg.nnodes, 
          local_infinity,
          roundNumber,
          lastRoundNumber,
          ctx->bcData_minDistance.data.gpu_wr_ptr(),
          ctx->bcData_shortPathCount.data.gpu_wr_ptr(),
          ctx->bcData_dependencyValue.data.gpu_wr_ptr(),
          // TODO: WESTON: hash map info?
          ctx->roundIndexToSend.data.gpu_wr_ptr(),
          ctx->mrbc_tree.data.gpu_wr_ptr());

  hipDeviceSynchronize();
  check_cuda_kernel;
}


// void BackPropOp(GNode dst, Graph& graph) {
//   NodeData& dst_data = graph.getData(dst);
//   unsigned i         = dst_data.roundIndexToSend;
// 
//   if (i != infinity) {
//     uint32_t myDistance = dst_data.sourceData[i].minDistance;
// 
//     // calculate final dependency value
//     dst_data.sourceData[i].dependencyValue =
//       dst_data.sourceData[i].dependencyValue *
//         dst_data.sourceData[i].shortPathCount;
// 
//     // get the value to add to predecessors
//     float toAdd = ((float)1 + dst_data.sourceData[i].dependencyValue) /
//                   dst_data.sourceData[i].shortPathCount;
// 
//     for (auto inEdge : graph.edges(dst)) {
//       GNode src      = graph.getEdgeDst(inEdge);
//       auto& src_data = graph.getData(src);
//       uint32_t sourceDistance = src_data.sourceData[i].minDistance;
// 
//       // source nodes of this batch (i.e. distance 0) can be safely
//       // ignored
//       if (sourceDistance != 0) {
//         // determine if this source is a predecessor
//         if (myDistance == (sourceDistance + 1)) {
//           // add to dependency of predecessor using our finalized one
//           galois::atomicAdd(src_data.sourceData[i].dependencyValue, toAdd);
//         }
//       }   
//     }   
//   }
// }


__global__ void BackProp_kernel(
       CSRGraph graph, 
       unsigned int __begin, 
       unsigned int __end, 
       uint32_t local_infinity,
       uint32_t * p_minDistance,
       double   * p_shortPathCount,
       float    * p_dependencyValue,
       uint32_t * p_roundIndexToSend,
       MRBCTree_cuda * p_mrbc_tree)
{
    unsigned tid = TID_1D;
    unsigned nthreads = TOTAL_THREADS_1D;

    //const unsigned __kernel_tb_size = TB_SIZE;
    index_type dst_end;

    dst_end = __end;
    for (index_type dst = __begin + tid; dst < dst_end; dst += nthreads)
    {
      unsigned i = p_roundIndexToSend[dst];

      if (i != local_infinity) {
        uint32_t myDistance = p_minDistance[dst + (i * graph.nnodes)];

        p_dependencyValue[dst + (i * graph.nnodes)] = 
          p_dependencyValue[dst + (i * graph.nnodes)] 
            * p_shortPathCount[dst + (i * graph.nnodes)];

        float toAdd = ((float)1 + p_dependencyValue[dst + (i * graph.nnodes)]) /
          p_shortPathCount[dst + (i * graph.nnodes)];


        index_type current_edge_end = graph.getFirstEdge((dst) + 1);
        for (index_type current_edge = graph.getFirstEdge(dst); 
               current_edge < current_edge_end;   
               current_edge += 1)
        {   
          
          index_type src = graph.getAbsDestination(current_edge);
          uint32_t sourceDistance = p_minDistance[src + (i * graph.nnodes)];

          if (sourceDistance != 0) {
            if (myDistance == (sourceDistance + 1)) {
              atomicAdd(&p_dependencyValue[src + (i * graph.nnodes)], toAdd);
            }
          }

        }

      }
    }

}


void BackProp_cuda(
    const uint32_t & local_infinity, 
    struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;

  kernel_sizing(blocks, threads);
  
  BackProp_kernel <<<blocks, threads>>>(
          ctx->gg, 
          0, 
          ctx->gg.nnodes, 
          local_infinity,
          ctx->minDistance.data.gpu_wr_ptr(),
          ctx->shortPathCount.data.gpu_wr_ptr(),
          ctx->dependencyValue.data.gpu_wr_ptr(),
          ctx->roundIndexToSend.data.gpu_wr_ptr(),
          ctx->mrbc_tree.data.gpu_wr_ptr());

  hipDeviceSynchronize();
  check_cuda_kernel;
}
//  galois::do_all(
//       galois::iterate(masterNodes.begin(), masterNodes.end()),
//       [&](GNode node) {
//         NodeData& cur_data = graph.getData(node);
//
//         for (unsigned i = 0; i < numSourcesPerRound; i++) {
//           // exclude sources themselves from BC calculation
//           if (graph.getGID(node) != nodesToConsider[i]) {
//             cur_data.bc += cur_data.sourceData[i].dependencyValue;
//           }
//         }
//       },
//       galois::loopname(syncSubstrate->get_run_identifier("BC").c_str()),
//       galois::no_stats());



__global__ void BC_kernel(
       CSRGraph graph, 
       unsigned int __begin, 
       unsigned int __end, 
       unsigned int numSourcesPerRound,
       uint64_t *  cuda_nodes_to_consider,
       uint32_t * p_minDistance,
       double   * p_shortPathCount,
       float    * p_dependencyValue,
       uint32_t * p_roundIndexToSend,
       float* p_bc)
{
    unsigned tid = TID_1D;
    unsigned nthreads = TOTAL_THREADS_1D;

    //const unsigned __kernel_tb_size = TB_SIZE;
    index_type src_end;

    src_end = __end;
    for (index_type src = __begin + tid; src < src_end; src += nthreads)
    {
      for (unsigned i = 0; i < numSourcesPerRound; i++) {
        if (graph.node_data[src] != cuda_nodes_to_consider[i]) {
          p_bc[src] += p_dependencyValue[src + (i * graph.nnodes)];
        }
      }
    }

}


void BC_cuda(
    struct CUDA_Context*  ctx,
    const uint64_t* local_nodes_to_consider)
{
  dim3 blocks;
  dim3 threads;

  kernel_sizing(blocks, threads);

  // Make device vector for local_nodes_to_consider
  uint64_t* cuda_nodes_to_consider;
  hipMalloc((void**) &cuda_nodes_to_consider, ctx->vectorSize*sizeof(uint64_t));
  hipMemcpy(cuda_nodes_to_consider, local_nodes_to_consider, ctx->vectorSize*sizeof(uint64_t), hipMemcpyHostToDevice);
  
  BC_kernel <<<blocks, threads>>>(
          ctx->gg, 
          0, 
          ctx->gg.nnodes, 
          ctx->vectorSize,
          cuda_nodes_to_consider,
          ctx->minDistance.data.gpu_wr_ptr(),
          ctx->shortPathCount.data.gpu_wr_ptr(),
          ctx->dependencyValue.data.gpu_wr_ptr(),
          ctx->roundIndexToSend.data.gpu_wr_ptr(),
          ctx->bc.data.gpu_wr_ptr());

  hipDeviceSynchronize();
  check_cuda_kernel;
}


//  galois::do_all(galois::iterate(graph.masterNodesRange().begin(),
//                                  graph.masterNodesRange().end()),
//                  [&](auto src) {
//                    NodeData& sdata = graph.getData(src);
//
//                    DGA_max.update(sdata.bc);
//                    DGA_min.update(sdata.bc);
//                    DGA_sum += sdata.bc;
//                  },
//                  galois::no_stats(), galois::loopname("Sanity"));



void Sanity_cuda(struct CUDA_Context* ctx);



// TODO: WESTON: write cuda bitset class
void bitset_dependency_reset_cuda(struct CUDA_Context* ctx);
void bitset_minDistances_reset_cuda(struct CUDA_Context* ctx);
